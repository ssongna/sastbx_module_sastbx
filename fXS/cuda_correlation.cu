#include "hip/hip_runtime.h"
#include <sastbx/fXS/cuda_correlation.cuh>

namespace sastbx {
namespace fXS {

  const int blocks_per_grid = 128;
  const int threads_per_block = 1024;
  const int chunk_size = blocks_per_grid*threads_per_block;

  /* ==========================================================================
     Basic cumulative addition of complex arrays using streams
  */
  __global__ void add_images_kernel_streams
  (const hipDoubleComplex* image_chunk, const int begin, const int image_size,
   hipDoubleComplex* summed_image) {
    int chunk_index = blockDim.x * blockIdx.x + threadIdx.x;
    int image_index = begin + chunk_index;
    if (image_index < image_size) {
      summed_image[image_index] = hipCadd(summed_image[image_index],
                                         image_chunk[chunk_index]);
    }
  }

  scitbx::af::shared<std::complex<double> > cuda_add_images_streams
    (const scitbx::af::const_ref<std::complex<double> >& images,
     const int& image_size, const int& n_images, const int& gpu_id) {

    // start GPU
    cudaSafeCall( hipSetDevice(gpu_id) );

    // initialize timer
    // hipEvent_t start, stop;
    // cudaSafeCall( hipEventCreate(&start) );
    // cudaSafeCall( hipEventCreate(&stop) );
    // cudaSafeCall( hipEventRecord(start) );

    // allocate and initialize arrays
    hipDoubleComplex * h_images, * h_summed_image;
    hipDoubleComplex * d_images0, * d_images1, * d_summed_image;
    cudaSafeCall( hipHostAlloc( (void**)&h_images,
                                 images.size() * sizeof(hipDoubleComplex),
                                 hipHostMallocDefault ) );
    cudaSafeCall( hipHostAlloc( (void**)&h_summed_image,
                                 image_size * sizeof(hipDoubleComplex),
                                 hipHostMallocDefault ) );
    cudaSafeCall( hipMalloc( (void**)&d_images0,
                              chunk_size * sizeof(hipDoubleComplex) ) );
    cudaSafeCall( hipMalloc( (void**)&d_images1,
                              chunk_size * sizeof(hipDoubleComplex) ) );
    cudaSafeCall( hipMalloc( (void**)&d_summed_image,
                              image_size * sizeof(hipDoubleComplex) ) );
    for (int i=0; i<images.size(); i++) {
      h_images[i] = make_hipDoubleComplex(images[i].real(),images[i].imag());
    }
    for (int i=0; i<image_size; i++) {
      h_summed_image[i] = make_hipDoubleComplex(0.0,0.0);
    }
    cudaSafeCall( hipMemcpy( d_summed_image, h_summed_image,
                              image_size * sizeof(hipDoubleComplex),
                              hipMemcpyHostToDevice ) );

    // initialize streams
    hipStream_t s0, s1;
    cudaSafeCall( hipStreamCreate( &s0 ) );
    cudaSafeCall( hipStreamCreate( &s1 ) );

    int begin0, begin1, image_offset;
    for (int i=0; i<n_images; i++) {
      image_offset = i*image_size;
      for (int j=0; j<int(ceil(image_size/chunk_size)); j+=2) {
        // copy images one chunk at a time
        begin0 = j*chunk_size;
        begin1 = begin0 + chunk_size;
        cudaSafeCall( hipMemcpyAsync
                      ( d_images0,
                        h_images + image_offset + begin0,
                        chunk_size * sizeof(hipDoubleComplex),
                        hipMemcpyHostToDevice, s0 ) );
        cudaSafeCall( hipMemcpyAsync
                      ( d_images1,
                        h_images + image_offset + begin1,
                        chunk_size * sizeof(hipDoubleComplex),
                        hipMemcpyHostToDevice, s1 ) );
        // add to sum
        add_images_kernel_streams<<<blocks_per_grid,threads_per_block,0,s0>>>
          (d_images0,begin0,image_size,d_summed_image);
        add_images_kernel_streams<<<blocks_per_grid,threads_per_block,0,s1>>>
          (d_images1,begin1,image_size,d_summed_image);
      }
    }
    // copy result from GPU to host
    cudaSafeCall( hipStreamSynchronize( s0 ) );
    cudaSafeCall( hipStreamSynchronize( s1 ) );
    cudaSafeCall( hipMemcpy( h_summed_image, d_summed_image,
                              image_size * sizeof(hipDoubleComplex),
                              hipMemcpyDeviceToHost ) );
    scitbx::af::shared<std::complex<double> > summed_image(image_size);
    for (int i=0; i<image_size; i++) {
      summed_image[i] = std::complex<double>(hipCreal(h_summed_image[i]),
                                             hipCimag(h_summed_image[i]));
    }

    // clean up
    cudaSafeCall( hipHostFree( h_images ) );
    cudaSafeCall( hipHostFree( h_summed_image ) );
    cudaSafeCall( hipFree( d_images0 ) );
    cudaSafeCall( hipFree( d_images1 ) );
    cudaSafeCall( hipFree( d_summed_image ) );
    cudaSafeCall( hipStreamDestroy( s0 ) );
    cudaSafeCall( hipStreamDestroy( s1 ) );

    // end timer
    // cudaSafeCall( hipEventRecord(stop) );
    // cudaSafeCall( hipEventSynchronize(stop) );
    // float elapsedTime;
    // cudaSafeCall( hipEventElapsedTime(&elapsedTime,start,stop) );
    // std::cout << elapsedTime << "\n";
    // cudaSafeCall( hipEventDestroy(start) );
    // cudaSafeCall( hipEventDestroy(stop) );

    return summed_image;
  }

  /* ==========================================================================
     Fast implementation without streams (probably due to excessive transfers
     to and from global memory, and no copying of original data)
  */
  __global__ void add_images_kernel
  (const hipDoubleComplex* images, const int image_size, const int n_images,
   hipDoubleComplex* summed_image) {

    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // each thead sums over all images for one element
    if (i < image_size) {
      hipDoubleComplex sum = make_hipDoubleComplex(0.0,0.0);
      for (int c_i=0; c_i<n_images; c_i++) {
        sum = hipCadd(sum,images[c_i*image_size + i]);
      }

      // transfer sum to global
      summed_image[i] = sum;
    }
  }

  scitbx::af::shared<std::complex<double> > cuda_add_images
  (const scitbx::af::const_ref<std::complex<double> >& images,
   const int& image_size, const int& n_images, const int& gpu_id) {

    // start GPU
    cudaSafeCall( hipSetDevice(gpu_id) );

    // initialize timer
    // hipEvent_t start, stop;
    // cudaSafeCall( hipEventCreate(&start) );
    // cudaSafeCall( hipEventCreate(&stop) );
    // cudaSafeCall( hipEventRecord(start) );

    // allocate and initialize arrays
    hipDoubleComplex * h_images, * d_images;
    hipDoubleComplex * h_summed_image, * d_summed_image;
    h_images = (hipDoubleComplex*)&images[0];
    h_summed_image = new hipDoubleComplex[image_size];
    cudaSafeCall( hipMalloc( (void**)&d_images,
                              images.size() * sizeof(hipDoubleComplex) ) );
    cudaSafeCall( hipMalloc( (void**)&d_summed_image,
                              image_size * sizeof(hipDoubleComplex) ) );
    cudaSafeCall( hipMemcpy( d_images, h_images,
                              images.size() * sizeof(hipDoubleComplex),
                              hipMemcpyHostToDevice ) );

    // run kernel
    int bpg = (image_size + threads_per_block - 1)/threads_per_block;
    add_images_kernel<<<bpg,threads_per_block>>>
      (d_images,image_size,n_images,d_summed_image);

    // copy result from GPU
    cudaSafeCall( hipMemcpy( h_summed_image, d_summed_image,
                              image_size * sizeof(hipDoubleComplex),
                              hipMemcpyDeviceToHost ) );
    scitbx::af::shared<std::complex<double> > summed_image
      ((std::complex<double>*)&h_summed_image[0],
       (std::complex<double>*)&h_summed_image[0] + image_size);

    // clean up
    cudaSafeCall( hipFree( d_images ) );
    cudaSafeCall( hipFree( d_summed_image ) );

    // end timer
    // cudaSafeCall( hipEventRecord(stop) );
    // cudaSafeCall( hipEventSynchronize(stop) );
    // float elapsedTime;
    // cudaSafeCall( hipEventElapsedTime(&elapsedTime,start,stop) );
    // std::cout << elapsedTime << "\n";
    // cudaSafeCall( hipEventDestroy(start) );
    // cudaSafeCall( hipEventDestroy(stop) );

    return summed_image;
  }

}
}
