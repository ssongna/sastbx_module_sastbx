#include "hip/hip_runtime.h"
#include <sastbx/fXS/cuda_functions.cuh>

namespace sastbx {
namespace fXS {

  const int threads_per_block = 1024;

  // form factors
  const int max_types = 50;
  const int max_terms = 10;
  __device__ __constant__ float d_a[max_types * max_terms];
  __device__ __constant__ float d_b[max_types * max_terms];
  __device__ __constant__ float d_c[max_types];
  __device__ __constant__ int d_n_types;
  __device__ __constant__ int d_n_terms;

  // constants
  __device__ __constant__ float two_pi = float(2.0)*HIP_PI_F;
  const int padded_size = 16;
  __device__ __constant__ int d_padded_size = padded_size;

  /* ==========================================================================

     Memory properties for C2070, compute capability 2.0
     (CUDA C Programming Guide v 4.0, Appendix F)
     ---------------------------------------------------
     registers - register, r/w, "fast", per-multiprocessor   32K 4B registers
     local memory - r/w, "slow", per-thread                  512 KB
     __shared__ - shared memory, r/w, "fast", block-wide     48 KB
     __device__ - global memory, r/w, "slow", grid-wide,     6 GB
     __constant__ - constant memory, r, "fast", grid-wide    64 KB

     Shared memory is broken up into 32 banks and is interleaved into 32-bit
     words (4 bytes).  For example, an array of length 64 containing single
     precision values will have elements 0 and 32 in the same bank, 1 and 33
     in the same bank, etc.  To access shared memory with no conflicts (all
     threads get data with one read), each thread should read from a different
     bank, or have multiple threads read the same value in the same bank.  In
     the previous example, having all threads access element 0 or having each
     thread read a different element between 0 and 31, inclusive, will only
     require one read.  Accessing elements 0 and 32 will require two reads.

     Appendix F.4 describes the memory properties for compute capability 2.0
     devices in more detail and has figures for efficient memory access
     patterns.

     Basic approach
     --------------
     Each thread calculates the sum for one h, so each thread will
     independently loop over all atoms and put the sum into global memory

     All coordinates are loaded into global memory and then each thread copies
     sections into shared memory.  The kernel loops over all sections to sum
     over all atoms.  Rotation matrix/translation vectors pairs are also loaded
     and looped in the same manner.  Form factors are stored in constant memory

     The thread index is checked against the length of the array multiple times
     because all threads are used for reading atom data from global, but only
     threads whose index is less than the array length are needed for
     summation.  Additionaly, two __syncthreads() calls are required.  The
     first is to make sure all the atom data is copied into shared memory
     before any summation is started, and the second is to make sure all the
     summation is finished before the atom data in shared memory is replaced
     with new data.

     Data format for kernel
     ----------------------
     xyz = x_0 ... x_n y_0 ... y_n z_0 ... z_n
     solvent_weights = s_0 s_1 ... s_n
     h = h_0 ... h_n k_0 ... k_n l_0 ... l_n
     rt = r_00 ... r_08 t_00 ... t_02 ... r_10 ... t_n2
     a = a_00 a_01 a_02 ... a_n3 a_n4 a_n5
     b = ""
     c = c_0 c_1 ... c_n

     To facilitate coalesced reads from global memory, the data is grouped
     into sections.  For example, for xyz, all the x's come first, then all
     the y's, and lastly, all the z's.  When read from global memory, three
     coalesced reads will read in all the xyz's for a set of 32 atoms, one
     read from each section.  The size of the shared arrays is equal to the
     number of threads so that all threads will attempt to read from global
     memory.  There are checks against the actual length of available data.

     For the structue_factor_kernel, the general format of the loops is,

     -----------------------------
     | x_0 | x_1 | x_2 | x_3 | ...          xyz array in global memory
     -----------------------------
        |     |     |     |                 each thread stores one value into
        |     |     |     |                 shared memory
        V     V     V     V                 x[threadIdx.x] = xyz[current_atom];
     -----------------------------
     | x_0 | x_1 | x_2 | x_3 | ...          x array in shared memory
     -----------------------------
        |
        |-----|-----|-----|                 each thread reads one value
        V     V     V     V                 x_a = x[a];
     --------------------------------------------------------
     |each thread calculates its own sum with its registers |
     --------------------------------------------------------
        |     |     |     |
        |     |     |     |                 loop over all atoms
        V     V     V     V
     -----------------------------
     | r_0 | r_1 | r_2 | r_3 | ...          each thread copies its sums into
     -----------------------------          the structure factor arrays in
     -----------------------------          global memory
     | i_0 | i_1 | i_2 | i_3 | ...
     -----------------------------

     --------------------------------------------------------------------------
  */

  // kernel
  __global__ void structure_factor_kernel
  (const int* scattering_type, const float* xyz,
   const float* solvent_weights, const int n_xyz,
   const float* h, const int n_h,
   const float* rt, const int n_rt,
   float* sf_real, float* sf_imag) {

    int i = blockDim.x * blockIdx.x + threadIdx.x;

    float h_i, k_i, l_i, stol_sq;
    float f[max_types];
    if (i < n_h) {
      // read h from global memory (stored in registers)
      h_i = h[i];
      k_i = h[n_h + i];
      l_i = h[2*n_h + i];

      // calculate form factors (stored in local memory)
      // last form factor is always for boundary solvent layer
      stol_sq = float(0.25) * (h_i*h_i + k_i*k_i + l_i*l_i);
      for (int type=0; type<d_n_types; type++) {
        f[type] = 0.0;
        for (int term=0; term<d_n_terms; term++) {
          f[type] += d_a[type*d_n_terms + term] *
            __expf(-d_b[type*d_n_terms + term] * stol_sq);
        }
        f[type] += d_c[type];
      }
    }

    // copy atoms into shared memory one chunk at a time and sum
    // all threads are used for reading data
    // shared arrays can be allocated at kernel invocation, but it requires
    // partitioning a big array (implement later)
    __shared__ float x[threads_per_block];
    __shared__ float y[threads_per_block];
    __shared__ float z[threads_per_block];
    __shared__ float solvent[threads_per_block];
    __shared__ int s_type[threads_per_block];
    __shared__ float rot_trans[threads_per_block];
    float real_sum = 0.0;
    float imag_sum = 0.0;
    float s,c,ff,xx,yy,zz,x_a,y_a,z_a;
    int current_atom, current_rt, rt_offset;

    for (int atom=0; atom<n_xyz; atom += blockDim.x) {
      current_atom = atom + threadIdx.x;
      // coalesce reads using threads, but don't read past n_xyz
      // one read for each variable should fill chunk of 32 atoms
      // total length = # of threads/block
      if (current_atom < n_xyz) {
        x[threadIdx.x] = xyz[          current_atom];
        y[threadIdx.x] = xyz[n_xyz   + current_atom];
        z[threadIdx.x] = xyz[2*n_xyz + current_atom];
        solvent[threadIdx.x] = solvent_weights[current_atom];
        s_type[threadIdx.x] = scattering_type[current_atom];
      }

      // loop over all rotation/translation operators
      // one coalesced read will copy (# of threads)/(padded_size) rot/trans
      // since the number of threads is a multiple of 32, it will also always
      // be evenly divisible by padded_size
      for (int rt_i=0; rt_i<n_rt; rt_i += blockDim.x/d_padded_size) {
        current_rt = rt_i*d_padded_size + threadIdx.x;
        if (current_rt < n_rt*d_padded_size) {
          rot_trans[threadIdx.x] = rt[current_rt];
        }

        // wait for all data to be copied into shared memory
        __syncthreads();

        // then sum over all the atoms that are now available to all threads
        if (i < n_h) {
          for (int r=0; r<blockDim.x/d_padded_size; r++) {
            current_rt = rt_i + r;  // overall counter for rot/trans pairs
            if (current_rt < n_rt) {
              for (int a=0; a<blockDim.x; a++) {
                current_atom = atom + a;  // overall counter for atom number
                if (current_atom < n_xyz) {
                  x_a = x[a];  // transfer from shared memory to registers
                  y_a = y[a];  // might not be necessary due to cache
                  z_a = z[a];
                  rt_offset = r*d_padded_size;
                  // apply rotation and translation by expanding Rx + t
                  xx = (x_a*rot_trans[rt_offset    ] +
                        y_a*rot_trans[rt_offset + 1] +
                        z_a*rot_trans[rt_offset + 2] +
                        rot_trans[rt_offset + 9]);
                  yy = (x_a*rot_trans[rt_offset + 3] +
                        y_a*rot_trans[rt_offset + 4] +
                        z_a*rot_trans[rt_offset + 5] +
                        rot_trans[r*padded_size + 10]);;
                  zz = (x_a*rot_trans[rt_offset + 6] +
                        y_a*rot_trans[rt_offset + 7] +
                        z_a*rot_trans[rt_offset + 8] +
                        rot_trans[rt_offset + 11]);;
                  __sincosf(two_pi*(xx * h_i + yy * k_i + zz * l_i),&s,&c);
                  // bulk solvent correction in f
                  // boundary layer solvent scale in solvent
                  ff = f[s_type[a]] + solvent[a]*f[d_n_types-1];
                  real_sum += ff * c;
                  imag_sum += ff * s;
                }
              }
            }
          }
        }

        // wait before starting next chunk so data isn't changed for lagging threads
        __syncthreads();
      }
    }

    // transfer result to global memory
    if (i < n_h) {
      sf_real[i] = real_sum;
      sf_imag[i] = imag_sum;
    }
  }

  /* ==========================================================================
   */
  sastbx::fXS::cuda_direct_summation::cuda_direct_summation() {
    sf_size = 0;
  }

  sastbx::fXS::cuda_direct_summation::~cuda_direct_summation() {
    cudaSafeCall( hipFree(sf_real) );
    cudaSafeCall( hipFree(sf_imag) );
  }

  /* --------------------------------------------------------------------------
     reorganizes data and calls cuda
  */
  void sastbx::fXS::cuda_direct_summation::add
  (const scitbx::af::const_ref<std::string>& scatterers,
   const scitbx::af::const_ref<scitbx::vec3<double> >& xyz,
   const scitbx::af::const_ref<double>& solvent_weights,
   const scitbx::af::const_ref<scitbx::vec3<double> >& h,
   const scitbx::af::const_ref<double>& rotations,
   const scitbx::af::const_ref<scitbx::vec3<double> >& translations,
   const cctbx::xray::scattering_type_registry& registry) {

    // reorganize coordinates
    int n_xyz = xyz.size();
    int size_xyz = 3 * n_xyz;
    float* h_xyz = new float[size_xyz];
    for (int i=0; i<n_xyz; i++) {
      for (int j=0; j<3; j++) {
        h_xyz[j*n_xyz + i] = float(xyz[i][j]);
      }
    }

    // copy boundary layer solvent weights
    float* h_solvent = new float[n_xyz];
    for (int i=0; i<n_xyz; i++) {
      h_solvent[i] = float(solvent_weights[i]);
    }

    // reorganize h
    int n_h = h.size();
    int size_h = 3 * n_h;
    float* h_h = new float[size_h];
    for (int i=0; i<n_h; i++) {
      for (int j=0; j<3; j++) {
        h_h[j*n_h + i] = float(h[i][j]);
      }
    }

    // reorganize rotations and translations
    // each rotation/translation pair is combined and padded to take up
    // 64 bytes so that a coalesced read will read two pairs
    int n_rt = translations.size();
    int size_rt = padded_size * n_rt;
    float* h_rt = new float[size_rt];
    for (int i=0; i<n_rt; i++) {
      for (int j=0; j<9; j++) {
        h_rt[padded_size*i + j] = float(rotations[9*i + j]);
      }
      for (int j=0; j<3; j++) {
        h_rt[padded_size*i + j + 9] = float(translations[i][j]);
      }
    }

    // convert scattering types and form factors
    // add ordinary oxygen form factor at end for boundary layer solvent
    int* h_scattering_type = new int[n_xyz];
    for (int i=0; i<n_xyz; i++) {
      h_scattering_type[i] = registry.unique_index(scatterers[i]);
    }
    scitbx::af::shared<boost::optional
                       <cctbx::eltbx::xray_scattering::gaussian> >
      unique_gaussians = registry.unique_gaussians;
    int n_types = unique_gaussians.size() + 1;
    int n_terms = unique_gaussians[0].get().n_terms();
    int f_size = n_types * n_terms;
    float* h_a = new float[f_size];
    float* h_b = new float[f_size];
    float* h_c = new float[n_types];
    for (int i=0; i<f_size; i++) {
      h_a[i] = 0.0;
      h_b[i] = 0.0;
    }
    for (int i=0; i<n_types-1; i++) {
      for (int j=0; j<n_terms; j++) {
        h_a[i*n_terms + j] = unique_gaussians[i].get().array_of_a()[j];
        h_b[i*n_terms + j] = unique_gaussians[i].get().array_of_b()[j];
      }
      if (unique_gaussians[i].get().use_c()) {
        h_c[i] = unique_gaussians[i].get().c();
      }
      else {
        h_c[i] = float(0.0);
      }
    }
    // add form factor for boundary layer solvent (# of terms may be different)
    cctbx::eltbx::xray_scattering::gaussian hoh =
      cctbx::eltbx::xray_scattering::wk1995("O",true).fetch();
    for (int i=0; i<hoh.array_of_a().size(); i++){
      h_a[(n_types-1)*n_terms + i] = hoh.array_of_a()[i];
      h_b[(n_types-1)*n_terms + i] = hoh.array_of_b()[i];
    }
    if (hoh.use_c()) {
      h_c[n_types-1] = hoh.c();
    }
    else {
      h_c[n_types-1] = float(0.0);
    }

    // transfer data to global memory
    int* d_scattering_type;
    cudaSafeCall( hipMalloc((void**)&d_scattering_type,n_xyz*sizeof(int)) );
    cudaSafeCall( hipMemcpy(d_scattering_type,h_scattering_type,
                             n_xyz*sizeof(int),hipMemcpyHostToDevice) );
    float* d_xyz;
    cudaSafeCall( hipMalloc((void**)&d_xyz,size_xyz*sizeof(float)) );
    cudaSafeCall( hipMemcpy(d_xyz, h_xyz, size_xyz*sizeof(float),
                             hipMemcpyHostToDevice) );
    float* d_solvent;
    cudaSafeCall( hipMalloc((void**)&d_solvent,n_xyz*sizeof(float)) );
    cudaSafeCall( hipMemcpy(d_solvent,h_solvent,n_xyz*sizeof(float),
                             hipMemcpyHostToDevice) );
    float* d_rt;
    cudaSafeCall( hipMalloc((void**)&d_rt,size_rt*sizeof(float)) );
    cudaSafeCall( hipMemcpy(d_rt, h_rt, size_rt*sizeof(float),
                             hipMemcpyHostToDevice) );
    float* d_h;
    cudaSafeCall( hipMalloc((void**)&d_h,size_h*sizeof(float)) );
    cudaSafeCall( hipMemcpy(d_h, h_h, size_h*sizeof(float),
                             hipMemcpyHostToDevice) );
    // transfer data to constant memory
    // should combine d_n_types and d_n_terms into one transfer
    cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(d_a), h_a, f_size*sizeof(float)) );
    cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(d_b), h_b, f_size*sizeof(float)) );
    cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(d_c), h_c, n_types*sizeof(float)) );
    cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(d_n_types), &n_types, sizeof(int)) );
    cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(d_n_terms), &n_terms, sizeof(int)) );

    // allocate arrays for results if necessary
    if (sf_size == 0) {
      sf_size = n_h;
      cudaSafeCall( hipMalloc((void**)&sf_real,n_h*sizeof(float)) );
      cudaSafeCall( hipMalloc((void**)&sf_imag,n_h*sizeof(float)) );
    }
    else {
      assert(sf_size == n_h);
    }

    // run calculation
    int blocks_per_grid = (n_h + threads_per_block - 1)/threads_per_block;
    structure_factor_kernel<<<blocks_per_grid,threads_per_block>>>
      (d_scattering_type, d_xyz, d_solvent, n_xyz,
       d_h, n_h,
       d_rt, n_rt,
       sf_real, sf_imag);

    // clean up
    delete[] h_xyz;
    delete[] h_solvent;
    delete[] h_h;
    delete[] h_rt;
    delete[] h_scattering_type;
    delete[] h_a;
    delete[] h_b;
    delete[] h_c;
    cudaSafeCall( hipFree(d_h) );
    cudaSafeCall( hipFree(d_xyz) );
    cudaSafeCall( hipFree(d_solvent) );
    cudaSafeCall( hipFree(d_rt) );
    cudaSafeCall( hipFree(d_scattering_type) );
  }

  /* --------------------------------------------------------------------------
     return total sum
  */
  scitbx::af::shared<std::complex<double> >
  sastbx::fXS::cuda_direct_summation::get_sum() {
    scitbx::af::shared<std::complex<double> > sf(sf_size);
    if (sf_size != 0) {
      float* h_real = new float[sf_size];
      float* h_imag = new float[sf_size];
      cudaSafeCall( hipMemcpy(h_real,sf_real,sf_size*sizeof(float),
                               hipMemcpyDeviceToHost) );
      cudaSafeCall( hipMemcpy(h_imag,sf_imag,sf_size*sizeof(float),
                               hipMemcpyDeviceToHost) );
      for (int i=0; i<sf_size; i++) {
        sf[i] = std::complex<double>(double(h_real[i]),double(h_imag[i]));
      }
      delete[] h_real;
      delete[] h_imag;
    }
    return sf;
  }

}
}
